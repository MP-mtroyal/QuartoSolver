#include "hip/hip_runtime.h"
﻿#include "quartoCuda.cuh"
#include "cudaHelpers.cuh"
#include <stdio.h>

#define GAME_SIZE 10
#define BLOCK_SIZE 4
#define X_DIM_LIMIT 65536

// ======== Structs ================================
struct QuartoCuda {
    int board[16];
    int numSelected;
    char selected[16];
    int numAvaliable;
    char avaliable[16];
    int recCounter;
};

// ============= HELPERS ===========================
__device__ bool checkFeatureListCuda(int* feats, int numFeats);


__device__ void LoadGameCuda(QuartoCuda* game, char* data) {
    int dataPieceIndex = 7;
    int dataPieceMaskShift = 0;

    game->numSelected = 0;
    game->numAvaliable = 0;
    game->recCounter = 0;

    char mask1 = data[8]; // pieces 1-8
    char mask2 = data[9]; // pieces 9-16

    for (int i = 15; i >= 0; i--) {
        // Fill selected and avaliable with default values
        game->selected[i] = -1;
        game->avaliable[i] = -1;
        game->board[i] = -1;

        if (i < 8) {
            if (mask1 & (1 << (7 - i))) {
                int currPiece = (data[dataPieceIndex] & (15 << dataPieceMaskShift)) >> dataPieceMaskShift;
                game->board[i] = currPiece;
                dataPieceMaskShift = (dataPieceMaskShift + 4) % 8;
                if (dataPieceMaskShift == 0) {
                    dataPieceIndex -= 1;
                }
            }
        }
        else {
            if (mask2 & (1 << (15 - i))) {
                int currPiece = (data[dataPieceIndex] & (15 << dataPieceMaskShift)) >> dataPieceMaskShift;
                game->board[i] = currPiece;
                dataPieceMaskShift = (dataPieceMaskShift + 4) % 8;
                if (dataPieceMaskShift == 0) {
                    dataPieceIndex -= 1;
                }
            }
        }
    }

    for (int i = 0; i < 16; i++) {
        bool found = false;
        for (int n = 0; n < 16; n++) {
            if (game->board[n] == i) {
                found = true;
                break;
            }
        }
        if (!found) {
            game->avaliable[game->numAvaliable] = i;
            game->numAvaliable += 1;
        }
    }
}

__device__ bool checkWinCuda(QuartoCuda* game) {
    int featList[4];
    //Horizontal I think
    for (int i = 0; i < 16; i += 4) {
        if (checkFeatureListCuda(game->board + i, 4))
            return true;
    }
    //Vertical I think
    for (int i = 0; i < 4; i++) {
        for (int n = 0; n < 4; n++) {
            featList[n] = game->board[i + (4 * n)];
        }
        if (checkFeatureListCuda(featList, 4))
            return true;
    }
    //Diagonal
    for (int i = 0; i < 4; i++) {
        featList[i] = game->board[i + (i * 4)];
    }
    if (checkFeatureListCuda(featList, 4))
        return true;
    // Anti Diagonal
    for (int i = 0; i < 4; i++) {
        featList[i] = game->board[i + ((3 - i) * 4)];
    }
    if (checkFeatureListCuda(featList, 4))
        return true;
    return false;
}

__device__ bool checkFeatureListCuda(int* feats, int numFeats) {
    char andCmp = 15;
    char orCmp = 0;
    for (int i = 0; i < numFeats; i++) {
        if (feats[i] < 0)
            return false;
        andCmp = andCmp & feats[i];
        orCmp = orCmp | feats[i];
    }
    return andCmp != 0 || 15 - orCmp != 0;
}

__device__ void removeFromArrayCuda(char* arr, char val, int len) {
    bool found = false;
    for (int i = 0; i < len; i++) {
        if (!found) {
            if (arr[i] == val) {
                found = true;
            }
        }
        else {
            arr[i - 1] = arr[i];
        }
    }
    arr[len - 1] = -1;
}

__device__ void selectPieceCuda(QuartoCuda* game, int piece) {
    // Add piece to end of selected array
    game->selected[game->numSelected] = piece;
    game->numSelected++;
    // Remove piece from avaliable array
    removeFromArrayCuda(game->avaliable, piece, 16);
    game->numAvaliable--;
}


__device__ void deselectPieceCuda(QuartoCuda* game, int piece) {
    // Add piece to end of avaliable array
    game->avaliable[game->numAvaliable] = piece;
    game->numAvaliable++;
    // Remove piece from selected array
    removeFromArrayCuda(game->selected, piece, game->numSelected);
    game->numSelected--;
}

__device__ void placePieceCuda(QuartoCuda* game, int piece, int place) {
    if (game->board[place] >= 0) {
        return;
    }
    game->board[place] = piece;

    removeFromArrayCuda(game->selected, piece, game->numSelected);
    game->numSelected--;
}

__device__ void removePieceCuda(QuartoCuda* game, int piece, int place) {
    game->board[place] = -1;

    game->selected[game->numSelected] = piece;
    game->numSelected++;
}

__device__ void copyStrCuda(char* srcStr, char* dstStr) {
    for (int i = 0; i < 1000; i++) { // dummy counter, just to prevent an infinite loop
        dstStr[i] = srcStr[i];
        if (srcStr[i] == '\0')
            return;
    }
}

__device__ void printGameCuda(QuartoCuda* game) {
    for (int i = 0; i < 16; i++) {
        if (game->board[i] < 0) {
            printf("    ");
        }
        else {
            for (int n = 8; n > 0; n /= 2) {
                if (game->board[i] & n)
                    printf("1");
                else
                    printf("0");
            }
        }
        if (i != 15) {
            if (i % 4 == 3)
                printf("\n----|----|----|----\n");
            else
                printf("|");
        }
    }
    printf("\n");
    // print selected
    printf("Selected: ");
    for (int i = 0; i < game->numSelected; i++) {
        for (int n = 3; n >= 0; n--) {
            if (game->selected[i] & (1 << n)) {
                printf("1");
            }
            else {
                printf("0");
            }
        }
        if (i != game->numSelected - 1) {
            printf(", ");
        }
    }
    printf("\n");

    // print Avaliable
    printf("Avaliable: ");
    for (int i = 0; i < game->numAvaliable; i++) {
        for (int n = 3; n >= 0; n--) {
            if (game->avaliable[i] & (1 << n)) {
                printf("1");
            }
            else {
                printf("0");
            }
        }
        if (i != game->numAvaliable - 1) {
            printf(", ");
        }
    }
    printf("\n\n");
}
// =================================================

__device__ int solveGameRec(
    QuartoCuda* game,
    char* sol,
    int solIndex,
    int depth,
    bool turn,
    bool placingPiece
    ) {
    // ====== Check if game is finished
    
    if (game->recCounter >= 500000) { // Only finds shallow depth solutions
        sol[solIndex] = '-';
        sol[solIndex+1] = '\0';
        return -1;
    }
    game->recCounter++;
    bool isFinal = checkWinCuda(game);

    if (isFinal || game->numAvaliable <= 0) {
        //printf("------------ At Bottom ----------------\n");
        int score = 0;
        if (isFinal) {
            if (turn) { score = 1; }
            else { score = -1; }
        }

        sol[solIndex + 1] = '\0'; // Null termination
        sol[solIndex] = (char)(score + 49); //arbitrary ascii offset
        return score;
    }

    int bestScore = -2;
    int score;
    char bestSol[22];

    // Place Piece
    if (placingPiece) {
        int bestSquare;
        int avalSquares[16];
        int numSquares = 0;
        int currPiece = game->selected[0];
        for (int i = 0; i < 16; i++) {
            if (game->board[i] < 0) {
                avalSquares[numSquares] = i;
                numSquares++;
            }
        }
        for (int i = 0; i < numSquares; i++) {
            placePieceCuda(game, currPiece, avalSquares[i]);
            score = solveGameRec(game, sol, solIndex + 1, depth + 1, turn, false);
            removePieceCuda(game, currPiece, avalSquares[i]);

            if (!turn)
                score *= -1;

            if (score > bestScore) {
                bestScore = score;
                bestSquare = avalSquares[i];
                copyStrCuda(sol + solIndex + 1, bestSol);
            }
            if (bestScore > 0) {
                break;
            }
        }
        if (bestScore < -1) { // should only occur in error, but there are error boards
            for (int n = 0; n < 3; n++) {
                sol[solIndex + n] = 'X';
            }
            return -2;
        }
        copyStrCuda(bestSol, sol + solIndex + 1);
        int squareIndex = ((bestSquare % 4) << 2) + (bestSquare / 4);
        sol[solIndex] = (char)(squareIndex + 64);
        return bestScore;

    }

    // Select Piece

    else {
        int piece;
        int bestPiece = -1;
        // Store avaliable pieces in another arrary to avoid reordering on select/deselect
        char currAvaliable[16];
        for (int i = 0; i < game->numAvaliable; i++) {
            currAvaliable[i] = game->avaliable[i];
        }

        for (int i = 0; i < game->numAvaliable; i++) {
            piece = currAvaliable[i];
            selectPieceCuda(game, piece);
            score = solveGameRec(game, sol, solIndex + 1, depth + 1, !turn, true);
            deselectPieceCuda(game, piece);

            if (turn)
                score *= -1;

            if (score > bestScore) {
                bestScore = score;
                bestPiece = piece;
                copyStrCuda(sol + solIndex + 1, bestSol);
            }
            if (bestScore > 0) {
                break;
            }

        }
        if (bestScore < -1) { // should only occur in error, but there are error boards
            for (int n = 0; n < 3; n++) {
                sol[solIndex + n] = 'X';
            }
            return -2;
        }
        copyStrCuda(bestSol, sol + solIndex + 1);
        sol[solIndex] = (char)(bestPiece + 64);
        return bestScore;
    }
}

__global__ void solveGameKernel(
    char* games, 
    char* sols, 
    int numGames, 
    int solSize
)
{
    int xOffset = blockIdx.x * blockDim.x + threadIdx.x;
    int yOffset = blockIdx.y * blockDim.y + threadIdx.y;

    int index = yOffset * X_DIM_LIMIT + xOffset;

    if (index < numGames) {
        //printf("Starting problem: %d\n", index);
        QuartoCuda game;
        LoadGameCuda(&game, games + index * GAME_SIZE);
        solveGameRec(
            &game,
            sols,
            solSize * index,
            0,
            true,
            false
        );
        if (index % 100 == 0) { // Only print 1 in 100 to make it more readable
            //atomicAdd(successCounter, 100);
            printf("Completed Problem: %d\t in %d\t steps\n", index, game.recCounter);
        }
    }
}

void solveGamesCuda(
    char* loadedGames, 
    char* solutions,
    int numGames,
    int solSize
)
{
    char* cudaGames;
    char* cudaSol;
    int* successCounter;
    hipMalloc(&cudaGames, numGames * GAME_SIZE * sizeof(char));
    hipMalloc(&cudaSol, numGames * solSize * sizeof(char));
    // Increase stack size to ensure recursion can work
    // I needed 13.5 Gb VRAM to run a stack size of 65536
    // 32768 worked too, just using 2^16 because I have the VRAM
    size_t newStackSize = 32768;
    hipDeviceSetLimit(hipLimitStackSize, newStackSize);

    hipMemcpy(
        cudaGames,
        loadedGames,
        numGames * GAME_SIZE * sizeof(char),
        hipMemcpyHostToDevice
    );

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(
        (X_DIM_LIMIT + BLOCK_SIZE - 1) / BLOCK_SIZE, 
        (numGames + X_DIM_LIMIT * BLOCK_SIZE - 1) / (X_DIM_LIMIT * BLOCK_SIZE)
    );

    solveGameKernel << <gridDim, blockDim >> > (
        cudaGames,
        cudaSol,
        numGames,
        solSize
    );

    hipDeviceSynchronize();

    printf("Copying Solutions\n");
    hipMemcpy(
        solutions,
        cudaSol,
        numGames * solSize * sizeof(char),
        hipMemcpyDeviceToHost
    );
    printf("Finished Copying\n");

    hipFree(cudaGames);
    hipFree(cudaSol);
}
